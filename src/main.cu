
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <time.h>

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void dijkstra(int N, int *hasil_gabung, int *graph)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int src = index; src < N; src += stride){
    const int N_const = N;
    int dist[110]; // Ganti ini juga sesuai dengan nilai N 
    int sptSet[110]; // Ganti ini juga sesuai dengan nilai N

    for (int i = 0; i < N; i++) 
      dist[i] = INT_MAX, sptSet[i] = 0; 

    dist[src] = 0; 

    for (int count = 0; count < N - 1; count++) { 
      int min = INT_MAX, min_index; 
      for (int v = 0; v < N; v++) 
        if (sptSet[v] == 0 && dist[v] <= min) min = dist[v], min_index = v;
      int u = min_index; 
  
      sptSet[u] = 1; 

      for (int v = 0; v < N; v++) 
        if (!sptSet[v] && graph[u*N+v] && dist[u] != INT_MAX 
          && dist[u] + graph[u*N+v] < dist[v]) 
          dist[v] = dist[u] + graph[u*N+v]; 
    }

    for (int i=0; i<N; i++) {
          hasil_gabung[src*N+i] = dist[i];
      }
    }
}


int main(int argc, char** argv)
{
  int N = stoi(argv[1]);
  int *hasil_gabung;
  int *graph;

  hipMallocManaged(&hasil_gabung, N*N*sizeof(int));  
  hipMallocManaged(&graph, N*N*sizeof(int));

  srand(13517093);
  for(int i = 0;i<N;i++) {
      graph[i*N+i] = 0;
      for(int j = i+1;j<N;j++) {
          graph[i*N+j] = rand() % 23;
          if(graph[i*N+j] == 0) graph[i*N+j] = 1;
          graph[j*N+i] = graph[i*N+j];
      }
  }
  struct timeval start, end;
  // gettimeofday(&start, NULL);
  
  int blockSize = stoi(argv[2]);  
  int numBlocks = (N + blockSize - 1) / blockSize;

  clock_t tStart = clock();
  dijkstra<<<numBlocks , blockSize>>>(N, hasil_gabung, graph);

  hipDeviceSynchronize();

  //for (int i = 0;i < N;i++) {
    // for (int j = 0;j < N; j++) {
      // cout << graph[i*N+j];
       // if(j != N-1) {
         // cout << " ";
      // }
     //}
     //cout << endl;
  // }

  cout << "------DIJKSTRA-------" << endl;

   for (int i = 0;i < N;i++) {
     for (int j = 0;j < N; j++) {
       cout << hasil_gabung[i*N+j];
         if(j != N-1) {
            cout << " ";
          }
     }
     cout << endl;
  }
  // gettimeofday(&end, NULL);

  // double delta = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;
  // printf("Time execution : %lf", delta);
  printf("Time taken: %.2f microsekon\n", (double)(clock() - tStart)/CLOCKS_PER_SEC*1000000 );
  // https://www.geeksforgeeks.org/clock-function-in-c-c/
  // Free memory
  hipFree(hasil_gabung);
  hipFree(graph);
  
  return 0;
}