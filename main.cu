
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <time.h>

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void dijkstra(int N, int *hasil_gabung, int *graph)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int src = index; src < N; src += stride){
    const int N_const = N;
    int dist[106]; // Ganti ini juga sesuai dengan nilai N 
    int sptSet[106]; // Ganti ini juga sesuai dengan nilai N

    for (int i = 0; i < N; i++) 
      dist[i] = INT_MAX, sptSet[i] = 0; 

    dist[src] = 0; 

    for (int count = 0; count < N - 1; count++) { 
      int min = INT_MAX, min_index; 
      for (int v = 0; v < N; v++) 
        if (sptSet[v] == 0 && dist[v] <= min) min = dist[v], min_index = v;
      int u = min_index; 
  
      sptSet[u] = 1; 

      for (int v = 0; v < N; v++) 
        if (!sptSet[v] && graph[u*N+v] && dist[u] != INT_MAX 
          && dist[u] + graph[u*N+v] < dist[v]) 
          dist[v] = dist[u] + graph[u*N+v]; 
    }

    for (int i=0; i<N; i++) {
          hasil_gabung[src*N+i] = dist[i];
      }
    }
}

