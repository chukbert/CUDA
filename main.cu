#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

void Dijkstra(int src);
// Kernel function to add the elements of two arrays
__global__
void Dijkstra(int src) {
    int dist[V + 5];
	int sptSet[V + 5];

	for (int i = 0; i < V; i++) 
		dist[i] = INT_MAX, sptSet[i] = 0; 

	dist[src] = 0; 

	for (int count = 0; count < V - 1; count++) { 
		int min = INT_MAX, min_index; 
        for (int v = 0; v < V; v++) 
            if (sptSet[v] == 0 && dist[v] <= min) 
                min = dist[v], min_index = v;
        int u = min_index; 
 
		sptSet[u] = 1; 

		for (int v = 0; v < V; v++) 
			if (!sptSet[v] && graph[u][v] && dist[u] != INT_MAX 
				&& dist[u] + graph[u][v] < dist[v]) 
				dist[v] = dist[u] + graph[u][v]; 
	}

	for (int i=0; i<V; i++) {
        hasil_gabung[src][i] = dist[i];
    }
}


void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}


int main(void)
{
  int N = 1<<20;
  float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize map
    srand(13517093);
    for(int i = 0;i<V;i++) {
        graph[i][i] = 0;
        for(int j = i+1;j<V;j++) {
            graph[i][j] = rand() % 23;
            if(graph[i][j] == 0) graph[i][j] = 1;
            graph[j][i] = graph[i][j];
        }
    }


  // Run kernel on 1M elements on the GPU
    dijkstra<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}